#include "api.hpp"

UTILS_BEGIN

void cuda::mallocDevice(void **ptr, size_t size){
    hipMalloc(ptr, size);
}

void cuda::freeDevice(void *ptr){
    hipFree(ptr);
}

void cuda::sync(){
    hipDeviceSynchronize();
}

void cuda::h2d(void *dst, void *src, size_t size){
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void cuda::d2h(void *dst, void *src, size_t size){
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void cuda::d2d(void *dst, void *src, size_t size){
    hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}

void cuda::err(const char* file, int line){
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        std::cout << "[CUDA ERROR] : FILE [" << file << "] LINE[" << line << "] " << hipGetErrorString(err) << std::endl;
        assert(err == hipSuccess);
        exit(1);
    }
}

UTILS_END