#include "api.hpp"

namespace utils::cuda {
void setDevice(int deviceIdx) { hipSetDevice(deviceIdx); }

void mallocDevice(void **ptr, size_t size) { hipMalloc(ptr, size); }

void freeDevice(void *ptr) { hipFree(ptr); }

void sync() { hipDeviceSynchronize(); }

void h2d(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void d2h(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void d2d(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}

void err(const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "[CUDA ERROR] : FILE [" << file << "] LINE[" << line
                  << "] " << hipGetErrorString(err) << std::endl;
        assert(err == hipSuccess);
        exit(1);
    }
}
} // namespace utils::cuda
