/**
 * @file api.cu
 * @author Chunyuan Zhao (zhaochunyuan@stu.pku.edu.cn)
 * @brief 
 * @version 0.1
 * @date 2024-05-09
 * 
 * @copyright Copyright (c) 2024
 * 
 */
#include "api.hpp"

namespace utils::cuda {
void warmUpDevice(int deviceIdx){
    hipSetDevice(deviceIdx);
    hipFree(0);
}

void setDevice(int deviceIdx) { hipSetDevice(deviceIdx); }

void mallocDevice(void **ptr, size_t size) { hipMalloc(ptr, size); }

void freeDevice(void *ptr) { hipFree(ptr); }

void sync() { hipDeviceSynchronize(); }

void h2d(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void d2h(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void d2d(void *dst, void *src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}

void err(const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "[CUDA ERROR] : FILE [" << file << "] LINE[" << line
                  << "] " << hipGetErrorString(err) << std::endl;
        assert(err == hipSuccess);
        exit(1);
    }
}
} // namespace utils::cuda
